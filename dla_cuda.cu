#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdbool.h>

// --- Runtime Configuration Variables ---
int GRID_SIZE_RUNTIME = 200;
int NUM_PARTICLES_RUNTIME = 1000;
int MAX_STEPS_RUNTIME = 7000;
int INITIAL_FROZEN_POINTS_RUNTIME = 4; 
size_t MAX_RLE_LINE_BUFFER_SIZE_RUNTIME;


#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        fprintf(stderr, "CUDA error \"%s\" at %s:%d\n", hipGetErrorString(err_), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}

// --- Global Device Pointers ---
int* d_frozen_grid;
int* d_particle_grid;
int* d_contact_grid;
int* d_particles;
hiprandState* d_rand_states;
int* d_active_particle_count_atomic;
int* d_temp_particles;

// --- Host variable for active particles ---
int h_active_particles;

// --- Kernel Definitions ---
__global__ void init_frozen_grid_kernel(int* frozen_grid, int size_val) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < size_val && y < size_val) {
        frozen_grid[y * size_val + x] = 0;
    }
}

__global__ void init_particles_kernel(int* particles_arr, hiprandState* rand_states_arr, int num_particles_val, int size_val, unsigned long long seed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_particles_val) {
        hiprand_init(seed, i, 0, &rand_states_arr[i]);
        particles_arr[i * 2 + 0] = hiprand_uniform(&rand_states_arr[i]) * size_val; 
        particles_arr[i * 2 + 1] = hiprand_uniform(&rand_states_arr[i]) * size_val; 
        if (particles_arr[i * 2 + 0] >= size_val) particles_arr[i * 2 + 0] = size_val - 1;
        if (particles_arr[i * 2 + 1] >= size_val) particles_arr[i * 2 + 1] = size_val - 1;
    }
}

__global__ void clear_grid_kernel(int* grid, int total_size_val, int value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size_val) {
        grid[idx] = value;
    }
}

__global__ void random_walk_kernel(int* particles_arr, hiprandState* rand_states_arr, int current_active_particles, int size_val) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < current_active_particles) {
        hiprandState localState = rand_states_arr[i]; 
        int dx = (int)(floorf(hiprand_uniform(&localState) * 3.0f)) - 1;
        int dy = (int)(floorf(hiprand_uniform(&localState) * 3.0f)) - 1;
        int current_x = particles_arr[i * 2 + 0];
        int current_y = particles_arr[i * 2 + 1];
        current_x += dx; current_y += dy;
        if (current_x < 0) current_x = 0; if (current_x >= size_val) current_x = size_val - 1;
        if (current_y < 0) current_y = 0; if (current_y >= size_val) current_y = size_val - 1;
        particles_arr[i * 2 + 0] = current_x; particles_arr[i * 2 + 1] = current_y;
        rand_states_arr[i] = localState;
    }
}

__global__ void update_particle_grid_kernel(int* particles_arr, int* particle_grid_arr, int current_active_particles, int size_val) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < current_active_particles) {
        int x = particles_arr[i * 2 + 0]; int y = particles_arr[i * 2 + 1];
        particle_grid_arr[y * size_val + x] = 1; 
    }
}

__global__ void generate_contact_grid_kernel(int* frozen_grid_arr, int* contact_grid_arr, int size_val) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < size_val && y < size_val) {
        if (frozen_grid_arr[y * size_val + x] == 1) { 
            for (int dy_kernel = -1; dy_kernel <= 1; dy_kernel++) for (int dx_kernel = -1; dx_kernel <= 1; dx_kernel++) {
                int nx = x + dx_kernel; int ny = y + dy_kernel;
                if (nx >= 0 && nx < size_val && ny >= 0 && ny < size_val) {
                    atomicAdd(&contact_grid_arr[ny * size_val + nx], 1); 
                }
            }
        }
    }
}

__global__ void calculate_frozen_grid_kernel(int* frozen_grid_arr, int* particle_grid_arr, int* contact_grid_arr, int size_val) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < size_val && y < size_val) {
        if (contact_grid_arr[y * size_val + x] > 0 && particle_grid_arr[y * size_val + x] == 1) { 
            frozen_grid_arr[y * size_val + x] = 1; 
        }
    }
}

__global__ void remove_frozen_particles_kernel(
    int* current_particles_arr, int* new_particles_buffer_arr, int* frozen_grid_arr,
    int current_active_count, int* new_active_count_atomic_ptr, int size_val, int max_particles_allowed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < current_active_count) {
        int px = current_particles_arr[i * 2 + 0]; int py = current_particles_arr[i * 2 + 1];
        if (frozen_grid_arr[py * size_val + px] == 0) { 
            int write_idx = atomicAdd(new_active_count_atomic_ptr, 1); 
            if (write_idx < max_particles_allowed) { 
                new_particles_buffer_arr[write_idx * 2 + 0] = px;
                new_particles_buffer_arr[write_idx * 2 + 1] = py;
            }
        }
    }
}

void host_init_grids_specific_points(int size_val, int num_initial_points) {
    int val = 1;

    if (size_val > 0) { 
        if (num_initial_points == 1) {
            int mid_r = size_val / 2;
            int mid_c = size_val / 2;
            CUDA_CHECK(hipMemcpy(&d_frozen_grid[mid_r * size_val + mid_c], &val, sizeof(int), hipMemcpyHostToDevice));
        } else if (num_initial_points > 1) {
            for (int k = 0; k < num_initial_points; k++) {
                if (size_val == 0) break; 
                int r_idx = rand() % size_val; 
                int c_idx = rand() % size_val; 
                CUDA_CHECK(hipMemcpy(&d_frozen_grid[r_idx * size_val + c_idx], &val, sizeof(int), hipMemcpyHostToDevice));
            }
        }
    }
}

void record_state_rle(FILE* f, int step, int* h_frozen_grid_cpu, int* h_particle_grid_cpu) {
    char *line_buffer = (char*)malloc(MAX_RLE_LINE_BUFFER_SIZE_RUNTIME);
    if (!line_buffer) {
        fprintf(stderr, "CUDA RLE Error: Failed to allocate line_buffer.\n");
        return;
    }
    char* current_pos_in_line = line_buffer;
    size_t remaining_buffer_size = MAX_RLE_LINE_BUFFER_SIZE_RUNTIME;
    int written;

    written = snprintf(current_pos_in_line, remaining_buffer_size, "%d,", step);
     if (written < 0 || (size_t)written >= remaining_buffer_size) { 
        fprintf(stderr, "CUDA RLE Error: Buffer too small for step number.\n"); free(line_buffer); return; 
    }
    current_pos_in_line += written; remaining_buffer_size -= written;

    int prev = -1; int count = 0;
    for (int i = 0; i < GRID_SIZE_RUNTIME; i++) { 
        for (int j = 0; j < GRID_SIZE_RUNTIME; j++) {
            int val = 0;
            if (h_frozen_grid_cpu[i * GRID_SIZE_RUNTIME + j] == 1) val = 2;
            else if (h_particle_grid_cpu[i * GRID_SIZE_RUNTIME + j] == 1) val = 1;
            
            if (val == prev) { 
                count++; 
            } else {
                if (prev != -1) {
                    written = snprintf(current_pos_in_line, remaining_buffer_size, "%dx%d,", prev, count);
                    if (written < 0 || (size_t)written >= remaining_buffer_size) {
                        fprintf(stderr, "CUDA RLE Error: Buffer too small for segment.\n"); 
                        goto end_line_write_cuda_dynamic;
                    }
                    current_pos_in_line += written; remaining_buffer_size -= written;
                }
                prev = val; count = 1;
            }
        }
    }
end_line_write_cuda_dynamic:
    if (count > 0 && prev != -1) {
        written = snprintf(current_pos_in_line, remaining_buffer_size, "%dx%d\n", prev, count);
        if (written < 0 || (size_t)written >= remaining_buffer_size) {
             fprintf(stderr, "CUDA RLE Error: Buffer too small for last segment.\n");
        }
    } else if (prev == -1 && count == 0) { 
        written = snprintf(current_pos_in_line, remaining_buffer_size, "0x%ld\n", (long)GRID_SIZE_RUNTIME*GRID_SIZE_RUNTIME);
         if (written < 0 || (size_t)written >= remaining_buffer_size) {
              fprintf(stderr, "CUDA RLE Error: Buffer too small for empty grid segment.\n");
         }
    } else if (count > 0 && prev == -1) { 
         written = snprintf(current_pos_in_line, remaining_buffer_size, "0x%ld\n", (long)GRID_SIZE_RUNTIME*GRID_SIZE_RUNTIME); 
         if (written < 0 || (size_t)written >= remaining_buffer_size) {
              fprintf(stderr, "CUDA RLE Error: Buffer too small for fallback empty grid segment.\n");
         }
    }
    fputs(line_buffer, f); 
    free(line_buffer); 
}

void reinitialize_cuda_state(unsigned long long seed_param) { 
    h_active_particles = NUM_PARTICLES_RUNTIME; 

    dim3 threadsPerBlock2D(16, 16);
    dim3 numBlocks2D((GRID_SIZE_RUNTIME + threadsPerBlock2D.x - 1) / threadsPerBlock2D.x,
                     (GRID_SIZE_RUNTIME + threadsPerBlock2D.y - 1) / threadsPerBlock2D.y);
    int threadsPerBlock1D = 256;

    init_frozen_grid_kernel<<<numBlocks2D, threadsPerBlock2D>>>(d_frozen_grid, GRID_SIZE_RUNTIME);
    CUDA_CHECK(hipGetLastError()); 
    host_init_grids_specific_points(GRID_SIZE_RUNTIME, INITIAL_FROZEN_POINTS_RUNTIME); 

    int numBlocksParticles = (NUM_PARTICLES_RUNTIME + threadsPerBlock1D - 1) / threadsPerBlock1D;
    init_particles_kernel<<<numBlocksParticles, threadsPerBlock1D>>>(d_particles, d_rand_states, NUM_PARTICLES_RUNTIME, GRID_SIZE_RUNTIME, seed_param);
    CUDA_CHECK(hipGetLastError());

    int totalGridCells = GRID_SIZE_RUNTIME * GRID_SIZE_RUNTIME;
    int numBlocksGridClear = (totalGridCells + threadsPerBlock1D - 1) / threadsPerBlock1D;
    clear_grid_kernel<<<numBlocksGridClear, threadsPerBlock1D>>>(d_particle_grid, totalGridCells, 0);
    CUDA_CHECK(hipGetLastError());
    clear_grid_kernel<<<numBlocksGridClear, threadsPerBlock1D>>>(d_contact_grid, totalGridCells, 0);
    CUDA_CHECK(hipGetLastError());
    
    CUDA_CHECK(hipMemset(d_active_particle_count_atomic, 0, sizeof(int))); 
    CUDA_CHECK(hipDeviceSynchronize()); 
}

void run_simulation_cuda(bool enable_rle_recording, const char* output_filename,
                         int* h_frozen_grid_cpu_buf, int* h_particle_grid_cpu_buf, int* steps_executed) {
    FILE* f_rle = NULL;
    *steps_executed = 0; 

    if (enable_rle_recording) {
        if (output_filename != NULL && remove(output_filename) == 0) {
        }
        f_rle = fopen(output_filename, "w");
        if (f_rle == NULL) {
            fprintf(stderr, "CUDA RLE run: Cannot open output file: %s\n", output_filename);
            return; 
        }
        printf("CUDA RLE run: Output will be written to %s\n", output_filename);
    }

    dim3 threadsPerBlock2D(16, 16);
    dim3 numBlocks2D((GRID_SIZE_RUNTIME + threadsPerBlock2D.x - 1) / threadsPerBlock2D.x,
                     (GRID_SIZE_RUNTIME + threadsPerBlock2D.y - 1) / threadsPerBlock2D.y);
    int threadsPerBlock1D = 256;
    int totalGridCells = GRID_SIZE_RUNTIME * GRID_SIZE_RUNTIME;
    int numBlocksGridClear = (totalGridCells + threadsPerBlock1D - 1) / threadsPerBlock1D;

    for (int step = 0; step < MAX_STEPS_RUNTIME; step++) {
        if (h_active_particles == 0) {
            if (enable_rle_recording && f_rle != NULL){ 
                 printf("CUDA RLE run: No active particles left at step %d. Stopping.\n", step);
            }
            break; 
        }

        int numBlocksActiveParticles = (h_active_particles + threadsPerBlock1D - 1) / threadsPerBlock1D;

        if (h_active_particles > 0) {
            random_walk_kernel<<<numBlocksActiveParticles, threadsPerBlock1D>>>(d_particles, d_rand_states, h_active_particles, GRID_SIZE_RUNTIME);
            CUDA_CHECK(hipGetLastError()); 
        }

        clear_grid_kernel<<<numBlocksGridClear, threadsPerBlock1D>>>(d_particle_grid, totalGridCells, 0);
        CUDA_CHECK(hipGetLastError());
        
        if (h_active_particles > 0) {
            update_particle_grid_kernel<<<numBlocksActiveParticles, threadsPerBlock1D>>>(d_particles, d_particle_grid, h_active_particles, GRID_SIZE_RUNTIME);
            CUDA_CHECK(hipGetLastError());
        }

        clear_grid_kernel<<<numBlocksGridClear, threadsPerBlock1D>>>(d_contact_grid, totalGridCells, 0);
        CUDA_CHECK(hipGetLastError());
        
        generate_contact_grid_kernel<<<numBlocks2D, threadsPerBlock2D>>>(d_frozen_grid, d_contact_grid, GRID_SIZE_RUNTIME);
        CUDA_CHECK(hipGetLastError());
        
        calculate_frozen_grid_kernel<<<numBlocks2D, threadsPerBlock2D>>>(d_frozen_grid, d_particle_grid, d_contact_grid, GRID_SIZE_RUNTIME);
        CUDA_CHECK(hipGetLastError());

        if (enable_rle_recording && f_rle != NULL) {
            CUDA_CHECK(hipMemcpy(h_frozen_grid_cpu_buf, d_frozen_grid, (size_t)GRID_SIZE_RUNTIME * GRID_SIZE_RUNTIME * sizeof(int), hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(h_particle_grid_cpu_buf, d_particle_grid, (size_t)GRID_SIZE_RUNTIME * GRID_SIZE_RUNTIME * sizeof(int), hipMemcpyDeviceToHost));
            record_state_rle(f_rle, step, h_frozen_grid_cpu_buf, h_particle_grid_cpu_buf);
        }

        CUDA_CHECK(hipMemset(d_active_particle_count_atomic, 0, sizeof(int))); 
        if (h_active_particles > 0) {
            remove_frozen_particles_kernel<<<numBlocksActiveParticles, threadsPerBlock1D>>>(
                d_particles, d_temp_particles, d_frozen_grid,
                h_active_particles, d_active_particle_count_atomic, GRID_SIZE_RUNTIME, NUM_PARTICLES_RUNTIME);
            CUDA_CHECK(hipGetLastError());
        }
        
        CUDA_CHECK(hipMemcpy(&h_active_particles, d_active_particle_count_atomic, sizeof(int), hipMemcpyDeviceToHost)); 
        if (h_active_particles > 0) {
            size_t bytes_to_copy = (size_t)h_active_particles * 2 * sizeof(int);
            if (bytes_to_copy <= (size_t)NUM_PARTICLES_RUNTIME * 2 * sizeof(int)) {
                 CUDA_CHECK(hipMemcpy(d_particles, d_temp_particles, bytes_to_copy, hipMemcpyDeviceToDevice));
            } else {
                fprintf(stderr, "Error: Attempting to copy too many particles after compaction. Active: %d, Max: %d\n", h_active_particles, NUM_PARTICLES_RUNTIME);
                h_active_particles = 0; 
            }
        }
        (*steps_executed)++; 
    }

    if (!enable_rle_recording) {
        CUDA_CHECK(hipDeviceSynchronize()); 
    }

    if (enable_rle_recording && f_rle != NULL) {
        fclose(f_rle);
        printf("CUDA RLE run: Simulation data recording complete.\n");
    }
}

void print_cuda_usage(const char* prog_name) {
    fprintf(stderr, "Usage: %s <grid_size> <num_particles> <max_steps> <initial_frozen_points> <benchmark_runs> [generate_csv]\n", prog_name);
    fprintf(stderr, "  <grid_size>: Integer, size of one dimension of the square grid (e.g., 200).\n");
    fprintf(stderr, "  <num_particles>: Integer, initial number of moving particles (e.g., 1000).\n");
    fprintf(stderr, "  <max_steps>: Integer, maximum simulation steps (e.g., 7000).\n");
    fprintf(stderr, "  <initial_frozen_points>: Integer, number of initial frozen seed points (e.g., 0, 1, or more).\n");
    fprintf(stderr, "                           If 1, placed in the center. If >1, placed randomly.\n");
    fprintf(stderr, "  <benchmark_runs>: Integer, number of times to run the simulation for benchmarking (e.g., 5).\n");
    fprintf(stderr, "  [generate_csv]: Optional string 'generate_csv' to output RLE data to a file.\n");
    fprintf(stderr, "Example: %s 200 1000 7000 1 5 generate_csv (1 point in center)\n", prog_name);
    fprintf(stderr, "Example: %s 200 1000 7000 4 5 generate_csv (4 random points)\n", prog_name);
    fprintf(stderr, "Example (benchmark only): %s 200 1000 7000 4 5\n", prog_name);
}


int main(int argc, char *argv[]) { 
    unsigned long long seed = 42; 
    const char* cuda_output_filename = "dla_output_rle_cuda.csv";
    bool perform_rle_run = false;
    int num_benchmark_runs = 1;
    int cuda_rle_actual_steps = 0; 

    if (argc < 6 || argc > 7) { 
        print_cuda_usage(argv[0]);
        return 1;
    }

    GRID_SIZE_RUNTIME = atoi(argv[1]);
    NUM_PARTICLES_RUNTIME = atoi(argv[2]);
    MAX_STEPS_RUNTIME = atoi(argv[3]);
    INITIAL_FROZEN_POINTS_RUNTIME = atoi(argv[4]); 
    num_benchmark_runs = atoi(argv[5]);            

    if (GRID_SIZE_RUNTIME <= 0 || NUM_PARTICLES_RUNTIME <= 0 || MAX_STEPS_RUNTIME < 0 || 
        INITIAL_FROZEN_POINTS_RUNTIME < 0 || num_benchmark_runs <= 0) {
        fprintf(stderr, "Error: grid_size, num_particles, benchmark_runs must be positive.\n");
        fprintf(stderr, "       max_steps and initial_frozen_points must be non-negative.\n");
        print_cuda_usage(argv[0]);
        return 1;
    }
    MAX_RLE_LINE_BUFFER_SIZE_RUNTIME = (size_t)GRID_SIZE_RUNTIME * GRID_SIZE_RUNTIME * 8 + 100;


    if (argc == 7 && strcmp(argv[6], "generate_csv") == 0) { 
        perform_rle_run = true;
    }
    
    srand((unsigned int)seed); 

    printf("CUDA Simulation Configuration:\n");
    printf("  Grid Size: %d x %d\n", GRID_SIZE_RUNTIME, GRID_SIZE_RUNTIME);
    printf("  Number of Particles: %d\n", NUM_PARTICLES_RUNTIME);
    printf("  Max Steps: %d\n", MAX_STEPS_RUNTIME);
    printf("  Initial Frozen Points: %d (if 1, center; if >1, random)\n", INITIAL_FROZEN_POINTS_RUNTIME);
    printf("  Benchmark Runs: %d\n", num_benchmark_runs);
    printf("  Generate CSV: %s\n", perform_rle_run ? "Yes" : "No");
    printf("--------------------------------------------------\n");


    CUDA_CHECK(hipMalloc((void**)&d_frozen_grid, (size_t)GRID_SIZE_RUNTIME * GRID_SIZE_RUNTIME * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_particle_grid, (size_t)GRID_SIZE_RUNTIME * GRID_SIZE_RUNTIME * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_contact_grid, (size_t)GRID_SIZE_RUNTIME * GRID_SIZE_RUNTIME * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_particles, (size_t)NUM_PARTICLES_RUNTIME * 2 * sizeof(int))); 
    CUDA_CHECK(hipMalloc((void**)&d_rand_states, (size_t)NUM_PARTICLES_RUNTIME * sizeof(hiprandState)));
    CUDA_CHECK(hipMalloc((void**)&d_active_particle_count_atomic, sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_temp_particles, (size_t)NUM_PARTICLES_RUNTIME * 2 * sizeof(int)));

    int* h_frozen_grid_cpu = (int*)malloc((size_t)GRID_SIZE_RUNTIME * GRID_SIZE_RUNTIME * sizeof(int));
    int* h_particle_grid_cpu = (int*)malloc((size_t)GRID_SIZE_RUNTIME * GRID_SIZE_RUNTIME * sizeof(int));
    if (!h_frozen_grid_cpu || !h_particle_grid_cpu) {
        fprintf(stderr, "Failed to allocate host memory for RLE buffers.\n"); 
        hipFree(d_frozen_grid); hipFree(d_particle_grid); hipFree(d_contact_grid);
        hipFree(d_particles); hipFree(d_rand_states);
        hipFree(d_active_particle_count_atomic); hipFree(d_temp_particles);
        return 1;
    }

    float total_milliseconds = 0;
    long long total_benchmark_steps_cuda = 0; // Use long long for total steps
    printf("\nStarting CUDA Benchmark Phase (%d timed runs)\n", num_benchmark_runs);
    int cuda_benchmark_iteration_steps = 0;

    for (int i = 0; i < num_benchmark_runs; ++i) {
        srand((unsigned int)seed + i); 
        reinitialize_cuda_state(seed + i); 

        hipEvent_t start_event, stop_event;
        CUDA_CHECK(hipEventCreate(&start_event));
        CUDA_CHECK(hipEventCreate(&stop_event));
        
        CUDA_CHECK(hipDeviceSynchronize()); 
        CUDA_CHECK(hipEventRecord(start_event, 0));

        run_simulation_cuda(false, NULL, h_frozen_grid_cpu, h_particle_grid_cpu, &cuda_benchmark_iteration_steps);

        CUDA_CHECK(hipEventRecord(stop_event, 0));
        CUDA_CHECK(hipEventSynchronize(stop_event)); 
        float current_run_ms = 0;
        CUDA_CHECK(hipEventElapsedTime(&current_run_ms, start_event, stop_event));
        total_milliseconds += current_run_ms;
        total_benchmark_steps_cuda += cuda_benchmark_iteration_steps;
        
        double time_per_step_ms = 0;
        if (cuda_benchmark_iteration_steps > 0) {
            time_per_step_ms = (double)current_run_ms / cuda_benchmark_iteration_steps;
        }

        printf("CUDA Benchmark Run %d/%d: %d steps, %f ms total, %e ms/step (%e seconds/step).\n", 
               i + 1, num_benchmark_runs, cuda_benchmark_iteration_steps, current_run_ms, 
               time_per_step_ms, time_per_step_ms / 1000.0);

        CUDA_CHECK(hipEventDestroy(start_event));
        CUDA_CHECK(hipEventDestroy(stop_event));
    }
    if (num_benchmark_runs > 0) {
        float average_total_ms = total_milliseconds / num_benchmark_runs;
        printf("CUDA Average Total Benchmark Time (over %d runs): %f ms (%f seconds).\n", 
               num_benchmark_runs, average_total_ms, average_total_ms / 1000.0f);
        if (total_benchmark_steps_cuda > 0) {
            double average_time_per_step_ms = (double)total_milliseconds / total_benchmark_steps_cuda;
            printf("CUDA Average Benchmark Time Per Step (over %lld steps): %e ms/step (%e seconds/step).\n", 
                   total_benchmark_steps_cuda, average_time_per_step_ms, average_time_per_step_ms / 1000.0);
        } else {
            printf("CUDA Average Benchmark Time Per Step: N/A (0 steps executed).\n");
        }
    }
    printf("--------------------------------------------------\n");
    

    if (perform_rle_run) {
        printf("\nStarting CUDA RLE Output Generation Run\n");
        srand((unsigned int)seed); 
        reinitialize_cuda_state(seed); 
        run_simulation_cuda(true, cuda_output_filename, h_frozen_grid_cpu, h_particle_grid_cpu, &cuda_rle_actual_steps);
        printf("--------------------------------------------------\n");
    }
    
    free(h_frozen_grid_cpu);
    free(h_particle_grid_cpu);
    hipFree(d_frozen_grid); hipFree(d_particle_grid); hipFree(d_contact_grid);
    hipFree(d_particles); hipFree(d_rand_states);
    hipFree(d_active_particle_count_atomic); hipFree(d_temp_particles);
    
    printf("\nCUDA simulation tasks complete.\n");
    if (perform_rle_run) {
        printf("The CUDA RLE simulation ran for %d steps.\n", cuda_rle_actual_steps);
    }
    return 0;
}
